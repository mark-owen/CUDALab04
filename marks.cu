#include "hip/hip_runtime.h"
// -*-c-*-
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/find.h>
#include <thrust/distance.h>

#define NUM_RECORDS 32768
#define THREADS_PER_BLOCK 256
//#define NUM_RECORDS 32
//#define THREADS_PER_BLOCK 8
#define NUMBER_LOOPS 1




//Exercise 1) CUDA Parallel reduction
__global__ void maximumMark_CUDA_kernel(float *d_marks, float *d_reduced_marks) {
  const int tx = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float sdata[THREADS_PER_BLOCK];
  
  //Load a single student mark
  sdata[tx] = d_marks[idx];
  
  //sync threads required to ensure all threads have finished writing
  __syncthreads();
  
  for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1){
    //Exercise 1.1) reduce two values per loop and write these back to shared memory	  
    
    
    // for each loop stride gives us the gap between the elements we want to read
    if( tx < stride) { // only do this for the number of threads needed
      float data0 = sdata[tx];
      float data1 = sdata[tx+stride];
      //printf("Stride %d tx %d data0 %f data1 %f\n", stride, tx, data0, data1);

      if(data0 > data1) sdata[tx] = data0;
      else sdata[tx] = data1;
    }  
    //sync threads required to ensure all threads have finished writing
    __syncthreads();
  }
  
  
  //Write the result to shared memory
  if (threadIdx.x == 0){
    d_reduced_marks[blockIdx.x] = sdata[0];
  }
}

void checkCUDAError(const char*);
void readmarks(float *marks);


void calculate_CPU(float*);
void maximumMark_CUDA(float*, float*);
void maximumMark_Thrust(float*);
int sortSplit_Thrust(float*);
int partition_Thrust(float *);



int main(void) {
	//function variables
	float *h_marks;
	float *h_marks_temp;
	hipEvent_t start, stop;
	float milliseconds = 0;
	int index = 0;

	//create some events for CUDA timing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//host allocation
	h_marks = (float*)malloc(sizeof(float)*NUM_RECORDS);
	h_marks_temp = (float*)malloc(sizeof(float)*NUM_RECORDS);

	//read file
	readmarks(h_marks);

	//find highest mark and marks higher than 90% (CPU)
	calculate_CPU(h_marks);

	//Exercise 1) find highest mark with shared memory
	maximumMark_CUDA(h_marks, h_marks_temp);

	//find highest mark (Thrust)
	maximumMark_Thrust(h_marks);

	//find number of marks greater than 90% using a sort and find
	hipEventRecord(start);
	for (int i = 0; i < NUMBER_LOOPS; i++){
		index = sortSplit_Thrust(h_marks);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("There are %d marks greater than 90%% (using sort+split)\n",index);
	printf("\t took %f ms to calculate\n", milliseconds);

	//find number of marks greater than 90% using partition (prefix sum and re-order)
	hipEventRecord(start);
	for (int i = 0; i < NUMBER_LOOPS; i++){
		index = partition_Thrust(h_marks);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("There are %d marks greater than 90%% (using partition)\n", index);
	printf("\t took %f ms to calculate\n", milliseconds);
	


	// Cleanup
	free(h_marks);
	free(h_marks_temp);



	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void readmarks(float *marks){
	FILE *f = NULL;


	f = fopen("marks.dat", "rb"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find marks.dat file \n");
		exit(1);
	}

	//read student marks
	if (fread(marks, sizeof(float), NUM_RECORDS, f) != NUM_RECORDS){
		fprintf(stderr, "Error: Unexpected end of file!\n");
		exit(1);
	}


	fclose(f);
}


void calculate_CPU(float *h_marks){
	unsigned int i;
	float max_mark;
	int count;

	max_mark = 0;
	count = 0;

	//iterate marks on CPU and record highest mark and student id
	for (i = 0; i < NUM_RECORDS; i++){
		float mark = h_marks[i];
		if (mark > max_mark){
			max_mark = mark;
		}
		if (mark > 90.0f)
			count++;
	}

	//output result
	printf("CPU: Highest mark recorded %f there are %d marks greater 90%%\n", max_mark, count);
}

void maximumMark_CUDA(float *h_marks, float *h_marks_temp){
	unsigned int i;
	float *d_marks;
	float *d_marks_temp;
	float max_mark;

	//device allocation
	hipMalloc((void**)&d_marks, sizeof(float)*NUM_RECORDS);
	hipMalloc((void**)&d_marks_temp, sizeof(float)*NUM_RECORDS);
	checkCUDAError("CUDA malloc");

	
	//memory copy marks to device
	hipMemcpy(d_marks, h_marks, sizeof(float)*NUM_RECORDS, hipMemcpyHostToDevice);
	checkCUDAError("CUDA: CUDA memcpy");

	//Call the shared memory reduction kernel
	dim3 blocksPerGrid(NUM_RECORDS / THREADS_PER_BLOCK, 1, 1);
	dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
	maximumMark_CUDA_kernel << <blocksPerGrid, threadsPerBlock >> >(d_marks, d_marks_temp);
	hipDeviceSynchronize();
	checkCUDAError("CUDA: CUDA kernel");


	//Exercise 1.2) copy result of block level reduction back to host and reduce these values serially
	max_mark = 0;
	// since we did the reduction at block level, we only need to get back an array with size of the number of blocks we have
	hipMemcpy(h_marks_temp, d_marks_temp, sizeof(float)*blocksPerGrid.x,hipMemcpyDeviceToHost);
	// final part is to find the highest value from the per-block values
	for(i=0; i<blocksPerGrid.x; ++i) {
	  if(h_marks_temp[i] > max_mark) max_mark = h_marks_temp[i];
	}

	//output result
	printf("CUDA: Highest mark recorded %f\n", max_mark);

	//cleanup
	hipFree(d_marks);
	hipFree(d_marks_temp);
	checkCUDAError("CUDA cleanup");
}

struct my_maximum
{
	__host__ __device__ float operator()(const float x, const float y)
	{
		return x < y ? y : x;
	}
};

//Exercise 1.3)
void maximumMark_Thrust(float *h_marks){
	float max_mark;

	//Exercise 1.3.1) create a thrust vector by copying from host array
	thrust::device_vector<float> d_marks(h_marks, h_marks+NUM_RECORDS);

	//Exercise 1.3.2) reduction using max operator
	max_mark = thrust::reduce( d_marks.begin(), d_marks.end(), 0.0, thrust::maximum<float>());

	printf("Thrust: Highest mark recorded %f\n", max_mark);

}



//Exercise 2.1
int sortSplit_Thrust(float *h_marks){
	thrust::device_vector<float>::iterator iter;

	// Exercise 2.1.1)

	//Exercise 2.1.2) sort the marks

	//Exercise 2.1.3) find if greater than 90%

	//Exercise 2.1.4) find index of first 90% mark
	return 0;

}

//Exercise 2.2)
int partition_Thrust(float * h_marks){
	thrust::device_vector<float>::iterator iter;
		
	//create a thrust vector

	//use a partition to apply a prefix sum and re-order

	//find index of first 90% mark
	return 0;

}


